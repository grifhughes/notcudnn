#include "layer.h"

struct layer *layer_create(int ro, int co, int bs)
{
        struct layer *l;

        hipMallocManaged(&l, sizeof(struct layer));
        if(!l) {
                fprintf(stderr, "alloc failed, exiting...");
                exit(-1);
        }
        hipMalloc((void **)&(l->w), sizeof(float) * ro * co);
        if(!l->w) {
                fprintf(stderr, "alloc failed, exiting...");
                exit(-1);
        }
        hipMalloc((void **)&(l->b), sizeof(float) * ro * bs);
        if(!l->b) {
                fprintf(stderr, "alloc failed, exiting...");
                exit(-1);
        }
        hipMalloc((void **)&(l->wi), sizeof(float) * ro * bs);
        if(!l->wi) {
                fprintf(stderr, "alloc failed, exiting...");
                exit(-1);
        }
        hipMalloc((void **)&(l->a), sizeof(float) * ro * bs);
        if(!l->a) {
                fprintf(stderr, "alloc failed, exiting...");
                exit(-1);
        }
        hipMalloc((void **)&(l->wu), sizeof(float) * ro * co);
        if(!l->wu) {
                fprintf(stderr, "alloc failed, exiting...");
                exit(-1);
        }
        hipMalloc((void **)&(l->bu), sizeof(float) * ro * bs);
        if(!l->bu) {
                fprintf(stderr, "alloc failed, exiting...");
                exit(-1);
        }

        hiprandGenerator_t gen;

        hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_XORWOW);
        if(!gen) {
                fprintf(stderr, "alloc failed, exiting...");
                exit(-1);
        }
        hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));
        hiprandGenerateUniform(gen, l->w, ro * co);
        hiprandGenerateUniform(gen, l->b, ro * bs);
        hiprandDestroyGenerator(gen);
        l->rows = ro;
        l->cols = co;
        return l;
}

void layer_destroy(struct layer *l)
{
        hipFree(l->bu);
        hipFree(l->wu);
        hipFree(l->a);
        hipFree(l->wi);
        hipFree(l->b);
        hipFree(l->w);
        hipFree(l);
}
